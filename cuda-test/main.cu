
#include <hip/hip_runtime.h>
#include <stdio.h>

int num_devices = 0;


/******************************************************************************
*
* Auxiliary routines
*
******************************************************************************/


#define error_check(error) do                 \
{                                             \
    cuda_assert((error), __FILE__, __LINE__); \
} while (0);

void cuda_assert(hipError_t code, const char *file, int line)
{
    if (code != hipSuccess) {
        fprintf(stderr,
            "[ERROR] Error code: %d Message: %s %s %d\n",
            code, hipGetErrorString(code), file, line);
        exit(code);
    }
}

void mem_ustring(char *s, long bytes)
{
    double usize = 0;
    if (bytes > 1E+9)
    {
        usize = bytes / 1E+9;
        sprintf(s, "%.2lf GB", usize);
    }
    else if (bytes > 1E+6)
    {
        usize = bytes / 1E+6;
        sprintf(s, "%.2lf MB", usize);
    }
    else if (bytes > 1E+3)
    {
        usize = bytes / 1E+3;
        sprintf(s, "%.2lf KB", usize);
    }
    else
    {
        usize = bytes;
        sprintf(s, "%lf Bytes", usize);
    }
}

void print_device_info(void)
{
    for (int n = 0; n < num_devices; n++)
    {
        char ustring[64];

        hipDeviceProp_t device_prop;
        hipGetDeviceProperties(&device_prop, n);

        size_t free_mem;;

        hipMemGetInfo(&free_mem, NULL);

        int device_threads = device_prop.multiProcessorCount * device_prop.maxThreadsPerMultiProcessor;

        printf("Device %d: %s\n", n, device_prop.name);
        printf("    Compute capability:      %d.%d\n", device_prop.major, device_prop.minor);
        printf("    Total number of threads: %d\n", device_threads);
        mem_ustring(ustring, device_prop.totalGlobalMem);
        printf("    Global memory size:      %s\n", ustring);
        mem_ustring(ustring, device_prop.sharedMemPerBlock);
        printf("    Shared memory size:      %s\n", ustring);
        mem_ustring(ustring, device_prop.totalConstMem);
        printf("    Constant memory size:    %s\n", ustring);
        mem_ustring(ustring, free_mem);
        printf("    Total free memory:       %s\n", ustring);
        printf("    Warp size:               %d\n", device_prop.warpSize);
    }
    printf("\n");
}


/******************************************************************************
*
* CUDA kernels
*
******************************************************************************/


__global__
void char_add(char *a, char *b)
{
    a[threadIdx.x] = b[threadIdx.x];
}


/******************************************************************************
*
* Device tests
*
******************************************************************************/


void run_device_test(void)
{
    for (int n = 0; n < num_devices; n++)
    {
        hipSetDevice(n);

        int string_size;

        char string[256] = "Hello world!";

        string_size = strlen(string) + 1;

        /* Allocate device */
        char *d_a, *d_b;
        hipMalloc((void **)&d_a, string_size * sizeof(char));
        hipMalloc((void **)&d_b, string_size * sizeof(char));

        /* Allocate host */
        char *a = (char *)calloc(string_size, sizeof(char));
        char *b = (char *)calloc(string_size, sizeof(char));

        strcpy(b, string);

        dim3 dim_block(string_size, 1);
        dim3 dim_grid(1);

        hipMemcpy((void *)d_b, (const void *)b, string_size * sizeof(char), hipMemcpyHostToDevice);

        char_add<<<dim_grid, dim_block>>>(d_a, d_b);
        error_check(hipPeekAtLastError());
        error_check(hipDeviceSynchronize());

        hipMemcpy((void *)a, (void *)d_a, string_size, hipMemcpyDeviceToHost);

        hipDeviceProp_t device_prop;
        hipGetDeviceProperties(&device_prop, n);

        if (strcmp(a, b) == 0)
            printf("[PASSED TEST] Device %d: %s\n", n, device_prop.name);
        else
            printf("[FAILED TEST] Device %d: %s\n", n, device_prop.name);

        hipFree(d_a);
        hipFree(d_b);
        free(a);
        free(b);
    }
    printf("\n");
}

int main(void)
{
    hipGetDeviceCount(&num_devices);
    if (num_devices > 0)
    {
        print_device_info();
        run_device_test();
    }
    else
    {
        printf("[ERROR] No CUDA devices found!\n");
    }

    return EXIT_SUCCESS;
}
